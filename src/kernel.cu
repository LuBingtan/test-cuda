#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel.h"  
__global__ void kernel()
{  
    printf("hello world from gpu!\n");  
}
__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
hipError_t cudaAdd(int* c,const int* a, const int* b, const unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if(cudaStatus != hipSuccess) {
		printf("set divice failed\n");
		goto ERROR;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if(cudaStatus != hipSuccess) {
		printf("cuda malloc a failed\n");
		goto ERROR;
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if(cudaStatus != hipSuccess) {
		printf("cuda malloc b failed\n");
		goto ERROR;
	}
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if(cudaStatus != hipSuccess) {
		printf("cuda malloc c failed\n");
		goto ERROR;
	}

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess) {
		printf("cuda memcpy a failed\n");
		goto ERROR;
	}
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess) {
		printf("cuda memcpy b failed\n");
		goto ERROR;
	}
	addKernel<<<1,size>>>(dev_c, dev_a, dev_b);
	
	cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess) {
		printf("addKernel failed: %s\n", hipGetErrorString(cudaStatus));
		goto ERROR;
	}
	cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess) {
		printf("device syn failed\n");
		goto ERROR;
	}
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess) {
		printf("cuda memcpy c failed\n");
		goto ERROR;
	}
ERROR:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return cudaStatus;
}
void gpuPrintf()
{
	kernel<<<1,10>>>();
	hipDeviceSynchronize();
}
